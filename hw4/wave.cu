/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define SIZE (MAXPOINTS+2)*sizeof(float)

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float values[MAXPOINTS+2], 	/* values at time t */
      oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
      newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
float *gvalues, *goldval, *gnewval;


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
                ,MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", 
                    MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

__global__ void VecAdd(float* A, int n) {
    /* Calculate initial values based on sine curve */
    for (int i = 1; i <= n; i++) {
        A[i] = sin(2.0 * PI * ((float)i - 1.0) / (float)(n - 1));
    } 
}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
    /* Calculate initial values based on sine curve */
    VecAdd<<<1, 1>>>(gvalues, tpoints);

    /* Initialize old values array */
    hipMemcpy(goldval, gvalues, SIZE, hipMemcpyDeviceToDevice);

    hipMemcpy(values, gvalues, SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(oldval, gvalues, SIZE, hipMemcpyDeviceToHost);

}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
    float dtime, c, dx, tau, sqtau;

    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;
    newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
    int i, j;

    /* Update values for each time step */
    for (i = 1; i<= nsteps; i++) {
        /* Update points along line for this time step */
        for (j = 1; j <= tpoints; j++) {
            /* global endpoints */
            if ((j == 1) || (j  == tpoints))
                newval[j] = 0.0;
            else
                do_math(j);
        }

        /* Update old values with new values */
        for (j = 1; j <= tpoints; j++) {
            oldval[j] = values[j];
            values[j] = newval[j];
        }
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
    int i;

    for (i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i%10 == 0)
            printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    if (argc < 3) {
        printf("Usage: ./wave <tpoints> <nsteps>\n");
        return 0;
    }
    sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
    check_param();

    // allocate GPU memory
    hipMalloc((float**)&gvalues, SIZE);
    hipMalloc((float**)&goldval, SIZE);
    hipMalloc((float**)&gnewval, SIZE);

    printf("Initializing points on the line...\n");
    init_line();
    printf("Updating all points for all time steps...\n");
    update();
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");

    // Free GPU memory
    hipFree(gnewval);
    hipFree(goldval);
    hipFree(gvalues);

    return 0;
}
