#include "hip/hip_runtime.h"
/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define SIZE (MAXPOINTS+22)*sizeof(float)

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float values[MAXPOINTS+22], 	/* values at time t */
      oldval[MAXPOINTS+22], 	/* values at time (t-dt) */
      newval[MAXPOINTS+22]; 	/* values at time (t+dt) */
float *gvalues, *goldval, *gnewval;

int grid, block;


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
                ,MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", 
                    MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

__global__ void VecAdd(float* A, int n) {
    /* Calculate initial values based on sine curve */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    A[i] = sin(2.0 * PI * ((float)i - 1.0) / (float)(n - 1));
}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
    /* Calculate initial values based on sine curve */
    VecAdd<<<grid, block>>>(gvalues, tpoints);
    hipDeviceSynchronize();

    /* Initialize old values array */
    hipMemcpy(goldval, gvalues, SIZE, hipMemcpyDeviceToDevice);
    //hipMemcpy(values, gvalues, SIZE, hipMemcpyDeviceToHost);
    //hipMemcpy(oldval, gvalues, SIZE, hipMemcpyDeviceToHost);
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__global__ void do_math(float* goldval, float* gvalues, float* gnewval, int n, float sqtau, int step) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int k = 0; k < step; ++k) {
        gnewval[i] = (2.0 * (1-sqtau) *gvalues[i]) - goldval[i];
        /* Update old values with new values */
        goldval[i] = gvalues[i];
        gvalues[i] = gnewval[i];
    }
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
    float dtime, c, dx, tau, sqtau;
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    /* Update values for each time step */
    /* Update points along line for this time step */
    do_math<<<grid, block>>>(goldval,  gvalues,  gnewval, tpoints, sqtau, nsteps);
    hipDeviceSynchronize();

    hipMemcpy(values, gvalues, SIZE, hipMemcpyDeviceToHost);
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
    int i;

    for (i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i%10 == 0)
            printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    if (argc < 3) {
        printf("Usage: ./wave <tpoints> <nsteps>\n");
        return 0;
    }
    sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
    check_param();

    // fill it up
    int left = tpoints%20;
    if (left) tpoints += (20 - left);
    grid = 20;
    block = tpoints/20;

    // allocate GPU memory
    hipMalloc((float**)&gvalues, SIZE);
    hipMalloc((float**)&goldval, SIZE);
    hipMalloc((float**)&gnewval, SIZE);

    printf("Initializing points on the line...\n");
    init_line();
    printf("Updating all points for all time steps...\n");
    update();

    // resume tpoints
    if (left) tpoints -= (20 - left);

    // result
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");

    // Free GPU memory
    hipFree(gnewval);
    hipFree(goldval);
    hipFree(gvalues);

    return 0;
}
